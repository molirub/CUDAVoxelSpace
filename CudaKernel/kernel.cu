#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
//#include "stdafx.h"
// Archivos de encabezado de Windows
#include <windows.h>
#include <stdio.h>
#include <math.h>

typedef struct camera_s
{
	float x;        // 512 // x position on the map
	float y;        // 800 // y position on the map
	float height;   // 78 // height of the camera
	float angle;    // 0 // direction of the camera
	float horizon;  // 100 // horizon position (look up and down)
	float distance; // 800   // distance of map
} camera_t;

bool is_initialized = false;

__global__
void voxel_kernel(int* img_width, int* img_height, int* map_width, int* map_height,
	              float* pleftx, float* plefty, float* dx, float* dy, float* z, float* height, float* scale_height, float* horizon,
	              unsigned char* rgb_colormap, unsigned char* heightmap, unsigned char* rgb_result)
{

	// Sacamos i (columna actual
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= *img_width) return;
	float pleftx_i = *pleftx + *dx * i;
	float plefty_i = *plefty + *dy * i;

	//Sacamos altura del pixel a dibujar
	//int offset_alturas = (int) (pleft[0] + 1024 * (int)pleft[1]);

	/* NOTA Sustituir 1023y 1024  por map_width*/
	int offset_alturas = (((int)(floorf(plefty_i)) & 1023) * 1024) + ((int)(floorf(pleftx_i)) & 1023);
	float height_on_screen = (((*height - heightmap[offset_alturas]) / (*z * *scale_height)) + *horizon);

	//Imprimimos colores hasta llegar al límite del buffer
	if (height_on_screen < 0) height_on_screen = 0;
	if (height_on_screen > *img_height) height_on_screen = *img_height;
	//Pintamos linea vertical
	for (int j = (int)(floorf(height_on_screen)); j < (int)(floorf(*img_height)); j++)
	{
		int index_rgb = ((*img_width * j + i) * 3);
		rgb_result[index_rgb + 0] = rgb_colormap[offset_alturas * 3 + 0];
		rgb_result[index_rgb + 1] = rgb_colormap[offset_alturas * 3 + 1];
		rgb_result[index_rgb + 2] = rgb_colormap[offset_alturas * 3 + 2];
	}
}

extern "C" {

	int map_size_rgb;
	int map_size;
	int size_rgb;

	// Allocate memory in device
	int* img_width_d;
	int* img_height_d;
	int* map_width_d;
	int* map_height_d;
	unsigned char* rgb_colormap_d;
	unsigned char* heightmap_d;
	unsigned char* rgb_result_d;

	float* pleftx_d;
	float* plefty_d;
	float* dx_d;
	float* dy_d;
	float* z_d;
	float* height_d;
	float* scale_height_d;
	float* horizon_d;


	/* This function is exported in DLL and calls cuda kernel*/
	int __declspec(dllexport) generate_voxel_image(int img_width, int img_height, int map_width, int map_height, camera_t camera, unsigned char* rgb_colormap, unsigned char* heightmap, unsigned char* rgb_result)
	{
		static bool is_initialized = false;
		static int map_size_rgb;
		static int map_size;
		static int size_rgb;

		static unsigned char* blue_sky;

		if (!is_initialized)
		{
			is_initialized = true;

			map_size_rgb = map_width * map_height * 3;
			map_size = map_width * map_height;
			size_rgb = img_width * img_height * 3;

			hipMalloc(&img_width_d, sizeof(int));
			hipMalloc(&img_height_d, sizeof(int));
			hipMalloc(&map_width_d, sizeof(int));
			hipMalloc(&map_height_d, sizeof(int));


			hipMalloc(&rgb_colormap_d, map_size_rgb * sizeof(char));
			hipMalloc(&heightmap_d, map_size * sizeof(char));
			hipMalloc(&rgb_result_d, size_rgb * sizeof(char));

			hipMalloc(&pleftx_d, sizeof(float));
			hipMalloc(&plefty_d, sizeof(float));
			hipMalloc(&dx_d, sizeof(float));
			hipMalloc(&dy_d, sizeof(float));
			hipMalloc(&z_d, sizeof(float));
			hipMalloc(&height_d, sizeof(float));
			hipMalloc(&scale_height_d, sizeof(float));
			hipMalloc(&horizon_d, sizeof(float));

			//Inicializar vector rgb a azulito para el cielo
			blue_sky = (unsigned char*)malloc(img_width * img_height * 3 * sizeof(unsigned char));
			for (int i = 0; i < img_width * img_height * 3; i += 3)
			{
				blue_sky[i] = 148;
				blue_sky[i + 1] = 209;
				blue_sky[i + 2] = 239;
			}

			// Copiamos memoria a device antes del bucle
			hipMemcpy(img_width_d, &img_width, sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(img_height_d, &img_height, sizeof(int), hipMemcpyHostToDevice);

			hipMemcpy(map_width_d, &map_width, sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(map_height_d, &map_height, sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(rgb_colormap_d, rgb_colormap, map_size_rgb * sizeof(char), hipMemcpyHostToDevice);
			hipMemcpy(heightmap_d, heightmap, map_size * sizeof(char), hipMemcpyHostToDevice);
		}

		// Perform some previous calculations
		// Copy data from host to device
		
		float p[2] = { camera.x, camera.y };
		//Altura cámara en bloques
		float height = camera.height;
		//Altura del horizonte 
		float horizon = camera.horizon;
		//Escalado de altura (cuanto más pequeño más exagerada la altura)
		float scale_height = 0.0015f;
		//Distancia máxima de renderizado en pixeles
		float distance = camera.distance;
		//Ángulo de la cámara en grados
		float phi = camera.angle;
		//phi = phi * 180.0f / 3.1415f;

		//Precalculamos los parámetros del ángulo de visión
		float sinphi = sin(phi);
		float cosphi = cos(phi);

		//Generación de cada una de la líneas de delante a detrás
		float z = 1.0f;
		float dz = 1.00f;
		
		hipMemcpy(rgb_result_d, blue_sky, size_rgb * sizeof(char), hipMemcpyHostToDevice);
		hipMemcpy(height_d, &height, sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(scale_height_d, &scale_height, sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(horizon_d, &horizon, sizeof(float), hipMemcpyHostToDevice);


		z = distance;
		while (z > 1)
		{

			//Cálculo de los extremos de las líneas para FOV 90º
			float pleft[2] = { (-cosphi * z - sinphi * z) + p[0],
								(sinphi * z - cosphi * z) + p[1] };
			float pright[2] = { (cosphi * z - sinphi * z) + p[0],
								(-sinphi * z - cosphi * z) + p[1] };

			//Segmentación líneas
			float dx = (pright[0] - pleft[0]) / (float)img_width;
			float dy = (pright[1] - pleft[1]) / (float)img_width;


			hipMemcpy(pleftx_d, &pleft[0], sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(plefty_d, &pleft[1], sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(dx_d, &dx, sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(dy_d, &dy, sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(z_d, &z, sizeof(float), hipMemcpyHostToDevice);

			// Launch kernel
			voxel_kernel <<< 64, 64 >>> (img_width_d, img_height_d, map_width_d, map_height_d, 
				                           pleftx_d, plefty_d, dx_d, dy_d, z_d, height_d, scale_height_d, horizon_d,
				                           rgb_colormap_d, heightmap_d, rgb_result_d);

			z -= dz;
			//dz -= 0.005; // Se pierde muhca resolucion
		}

		// Copiamos del device al host
		hipMemcpy(rgb_result, rgb_result_d, size_rgb * sizeof(char), hipMemcpyDeviceToHost);


		//hipFree(img_width_d);
		//hipFree(img_height_d);
		//hipFree(map_width_d);
		//hipFree(map_height_d);

		//hipFree(pleftx_d);
		//hipFree(plefty_d);
		//hipFree(dx_d);
		//hipFree(dy_d);
		//hipFree(z_d);
		//
		//hipFree(height_d);
		//hipFree(scale_height_d);
		//hipFree(horizon_d);

		//hipFree(rgb_colormap_d);
		//hipFree(heightmap_d);
		//hipFree(rgb_result_d);

		return 0;
	}
}

/*
//Posición cámara en 1024x1024
	//float p[2] = { 512.0f, 512.0f };
	float p[2] = { camera.x, camera.y };

	//Altura cámara en bloques
	//float height = 150.0f;
	float height = camera.height;

	//Altura del horizonte con respecto a IM_HEIGHT
	//float horizon = (float)IM_HEIGHT / 2.0f;
	float horizon = camera.horizon;

	//Escalado de altura (cuanto más pequeño más exagerada la altura)
	float scale_height = 0.0015f;

	//Distancia máxima de renderizado en pixeles
	//float distance = 4000.0f;
	float distance = camera.distance;

	//Ángulo de la cámara en grados
	//float phi = 0.0f;
	//phi = phi * 180.0f / 3.1415f;
	float phi = camera.angle;
	//phi = phi * 180.0f / 3.1415f;
	////////////////////////////////////////////

	//Inicializar vector rgb a gris para el cielo
	for (int i = 0; i < IM_HEIGHT * IM_WIDTH * 3; i+=3)
	{
		rgb[i] = 148;
		rgb[i+1] = 209;
		rgb[i+2] = 239;
	}

	//Precalculamos los parámetros del ángulo de visión
	float sinphi = sin(phi);
	float cosphi = cos(phi);

	//Inicializamos el vector de visibilidad para cada columna
	int ybuffer[IM_WIDTH];
	for (int i = 0; i < IM_WIDTH; i++)
	{
		ybuffer[i] = (int)IM_HEIGHT;
	}

	//Generación de cada una de la líneas de delante a detrás
	float z = 1.0f;
	float dz = 1.00f;
	while (z <= distance )
	{
		//Cálculo de los extremos de las líneas para FOV 90º
		float pleft[2] = { (-cosphi * z - sinphi * z) + p[0],
							(sinphi * z - cosphi * z) + p[1] };
		float pright[2] = { (cosphi * z - sinphi * z) + p[0],
							(-sinphi * z - cosphi * z) + p[1] };

		//Segmentación líneas
		float dx = (pright[0] - pleft[0]) / (float)IM_WIDTH;
		float dy = (pright[1] - pleft[1]) / (float)IM_WIDTH;

		//Wrapping
		//pleft[0] = fmodf(pleft[0], 1024.0f);
		//if (pleft[0] < 0.0f) pleft[0] += 1024.0f;
		//pleft[1] = fmodf(pleft[1], 1024.0f);
		//if (pleft[1] < 0.0f) pleft[1] += 1024.0f;

		// Truncado

		//Para cada pixel de la línea en la pantalla
		for (int i = 0; i < IM_WIDTH; i++)
		{
			//Sacamos altura del pixel a dibujar
			//int offset_alturas = (int) (pleft[0] + 1024 * (int)pleft[1]);

			int offset_alturas = (((int)floor(pleft[1]) & 1023) * 1024) + ((int)floor(pleft[0]) & 1023);
			float height_on_screen = (((height - valor_alturas[offset_alturas]) / (z * scale_height)) + horizon);

			//Imprimimos colores hasta llegar al límite del buffer
			if (height_on_screen < 0) height_on_screen = 0;
			if (height_on_screen > IM_HEIGHT) height_on_screen = IM_HEIGHT;
			for (int j = (int)floor(height_on_screen); j < (int)floor(ybuffer[i]); j++)
			{
				int index_rgb = ((IM_WIDTH * j + i) * 3);
				rgb[index_rgb + 0] = rgb_colores[offset_alturas * 3 + 0];
				rgb[index_rgb + 1] = rgb_colores[offset_alturas * 3 + 1];
				rgb[index_rgb + 2] = rgb_colores[offset_alturas * 3 + 2];
			}


			//Actualizamos límite del buffer
			if (height_on_screen <= ybuffer[i] && height_on_screen > 0)
				ybuffer[i] = height_on_screen;

			//Avanzamos un paso
			pleft[0] += dx;
			pleft[1] += dy;



			//Wrapping

			//pleft[0] = fmodf(pleft[0], 1024.0f);
			//if (pleft[0] < 0.0f) pleft[0] += 1024.0f;
			//pleft[1] = fmodf(pleft[1], 1024.0f);
			//if (pleft[1] < 0.0f) pleft[1] += 1024.0f;
		}

		z += dz;
		//dz += 0.005; // Se pierde muhca resolucion
	}

*/

BOOL APIENTRY DllMain(HMODULE hModule,
	DWORD  ul_reason_for_call,
	LPVOID lpReserved
)
{
	switch (ul_reason_for_call)
	{
	case DLL_PROCESS_ATTACH:
		break;
	case DLL_THREAD_ATTACH:
		break;
	case DLL_THREAD_DETACH:
		break;
	case DLL_PROCESS_DETACH:
		break;
	default:
		break;
	}

	return TRUE;
}